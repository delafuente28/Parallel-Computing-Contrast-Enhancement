#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
#include <assert.h>

// Histogram Kernel
__global__ void HistogramKernel(int * hist, unsigned char * img_in, int img_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= img_size) return; // If i value is greater than img_size, kernel execution will finish.
    
	// Stride is the total number of threads in grid
    int stride = blockDim.x * gridDim.x;
    
	// All threads handle blockDim.x * gridDim.x consecutive elements
    while (i < img_size) 
    {
		atomicAdd(&(hist[img_in[i]]), 1);
		i += stride;
	}
	// wait for all other threads in the block to finish
	__syncthreads();
}

// Get result image Kernel
__global__ void GetResultImageKernel(unsigned char * img_out_dev, unsigned char * img_in_dev, int * lut_dev, int img_size)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (i >= img_size) return; // If i value is greater than img_size, kernel execution will finish.

    // Stride is the total number of threads in grid
    int stride = blockDim.x * gridDim.x;
    
    while(i < img_size)
    {
        if(lut_dev[img_in_dev[i]] > 255)
        {
            img_out_dev[i] = 255;
        }
        
        if(lut_dev[img_in_dev[i]] <= 255)
        {
            img_out_dev[i] = (unsigned char)lut_dev[img_in_dev[i]];
        }
        i+=stride;
    }
    // Wait for all other threads in the block to finish
	__syncthreads();
}

void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin)
{
    int i;
    for (i = 0; i < nbr_bin; i++)
    {
        hist_out[i] = 0;
    }

    int * hist_out_buff;
    unsigned char * img_in_dev;

    // Allocate device memory
    size_t memSize = nbr_bin * sizeof(int);
    hipMalloc((void **)&hist_out_buff, memSize);
    hipMalloc((void **)&img_in_dev, img_size * sizeof(unsigned char));

    // Host to device memory copy
    hipMemcpy(img_in_dev, img_in, img_size, hipMemcpyHostToDevice);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpyHostToDevice");

    // Number of blocks - if the division is not exact, the resulting number is rounded up. 
    int numBlocks = (int)ceil(img_size / (float)numThreadsPerBlock);
    
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    HistogramKernel<<<dimGrid, dimBlock>>>(hist_out_buff, img_in_dev, img_size); //Kernel

    // Block until the device has completed
    hipDeviceSynchronize();

    // Check if kernel execution generated an error
    checkCUDAError("Kernel execution");

    // Device to host copy
    hipMemcpy(hist_out, hist_out_buff, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpyDeviceToHost");

    // Free device memory
    hipFree(hist_out_buff);
    hipFree(img_in_dev);
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, int * hist_in, int img_size, int nbr_bin)
{
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    
    while(min == 0)
    {
        min = hist_in[i++];
    }

    d = img_size - min;
   
    for(i = 0; i < nbr_bin; i++)
    {
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        
        if(lut[i] < 0)
        {
            lut[i] = 0;
        }        
    }

    unsigned char * img_out_buff;
    unsigned char * img_in_dev;
    int * lut_dev;

    // Allocate device memory
    hipMalloc((void **)&img_out_buff, img_size * sizeof(unsigned char));
    hipMalloc((void **)&img_in_dev, img_size * sizeof(unsigned char));
    hipMalloc((void **)&lut_dev, nbr_bin * sizeof(int));

    // Host to device memory copy
    hipMemcpy(img_in_dev, img_in, img_size, hipMemcpyHostToDevice); // Copy image to device
    hipMemcpy(lut_dev, lut, nbr_bin * sizeof(int), hipMemcpyHostToDevice); // Copy lut to device

    // Check for any CUDA errors
    checkCUDAError("hipMemcpyHostToDevice");

    // Number of blocks - if the division is not exact, the resulting number is rounded up. 
    int numBlocks = (int)ceil(img_size / (float)numThreadsPerBlock);

    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);

    /* Get the result image */
    GetResultImageKernel<<<dimGrid, dimBlock>>>(img_out_buff, img_in_dev, lut_dev, img_size); // Kernel
    
    // Block until the device has completed
    hipDeviceSynchronize();

    // Check for any CUDA errors
    checkCUDAError("Get Result Image Kernel Execution");

    // Device to host copy
    hipMemcpy(img_out, img_out_buff,  sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpyDeviceToHost");

    // Free device memory
    hipFree(img_in_dev);
    hipFree(lut_dev);
    hipFree(img_out_buff);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
        exit(-1);
    }
}